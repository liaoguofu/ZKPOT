#include "hip/hip_runtime.h"
#include "zksoftmax.cuh"
#include "zkfc.cuh"
#include "fr-tensor.cuh"
#include "proof.cuh"
#include "commitment.cuh"
#include "rescaling.cuh"
#include <string>


int main(int argc, char *argv[])
{    
 //   hipSetDevice(3);
    // 从命令行参数获取 vocab_size 和 seq_len
    uint vocab_size = std::stoi(argv[1]);
    string one_hot_file = argv[2];
    string input_file_name = argv[3];
    uint seq_len = std::stoi(argv[4]);
    uint embed_dim = std::stoi(argv[5]);
    string workdir = argv[6];
    string output_file_name = argv[7];

    auto lm_weight = create_weight(
        workdir + "/" + "lm_head.weight-pp.bin",
        workdir + "/" + "lm_head-weight-int.bin",
        workdir + "/" + "lm_head-weight-commitment.bin",
        seq_len, vocab_size
    );
    /*logits*/

    zkFC output_layer(4096, vocab_size, lm_weight.weight);

    FrTensor input = FrTensor::from_int_bin(input_file_name);
    Rescaling output_rescale(1 << 16);
    auto logits = output_layer(input);
    auto logits_ = output_rescale(logits);
    output_rescale.prove(logits, logits_);

    cout << "logits proof successfully verified!" << endl; 
    
    /*softmax*/
    auto padded_logits = pad_to_power_of_two(logits, seq_len, vocab_size);
    //cout << "padded_logits" <<padded_logits.size<< endl; 

    zkSoftmax softmax({1<<8, 1<<20, 1<<20}, 1, 0, 1UL<<32, {1<<18, 1<<22}, pow(2, ceil(log2(vocab_size))), seq_len, pow(2, ceil(log2(vocab_size))), 1);
    //zkSoftmax softmax({1<<8, 1<<8, 1<<20}, 1, 0, 1UL<<32, {1<<16, 1<<20}, vocab_size, seq_len, pow(2, ceil(log2(vocab_size))), 1);

    FrTensor shift(pow(2, ceil(log2(vocab_size)))), logits_shifted(seq_len * pow(2, ceil(log2(vocab_size))));    
   // FrTensor shift(vocab_size), logits_shifted(seq_len * vocab_size);    
    
    vector<FrTensor> logits_segments, Y_segments, m_segments;
    FrTensor Y = softmax.compute(padded_logits, shift, logits_shifted, logits_segments, Y_segments, m_segments);

    auto temp_rand = random_vec(3);
    vector<Polynomial> proof;
    cout << "Y" <<Y.size<< endl; 

    /*softmax.prove(Y, logits, shift, logits_shifted, logits_segments, Y_segments, m_segments, 
    random_vec(26), random_vec(26), temp_rand[0], temp_rand[1], temp_rand[2], proof);*/
    softmax.prove(Y, padded_logits, shift, logits_shifted, logits_segments, Y_segments, m_segments, 
    random_vec(26), random_vec(26), temp_rand[0], temp_rand[1], temp_rand[2], proof);
    cout << "softmax proof successfully verified!" << endl; 

 /*1.	交叉熵损失对outputnorm的梯度*/ 


    FrTensor one_hot = FrTensor::from_int_bin("one_hot_matrix.bin");

    auto one_hot_logits = pad_to_power_of_two(one_hot, seq_len, vocab_size);
    auto lm_weight_weight_padded = pad_to_power_of_two(lm_weight.weight, vocab_size, embed_dim);

    auto GD_logistin=Y-one_hot_logits;
    Y.~FrTensor();
    one_hot_logits.~FrTensor();

    //auto GD_outputnorm_out = FrTensor::matmul(Y, GD_logistin.transpose(seq_len, pow(2, ceil(log2(vocab_size)))), seq_len, pow(2, ceil(log2(vocab_size))), seq_len);
    auto GD_outputnorm_out = FrTensor::matmul( GD_logistin,lm_weight_weight_padded.transpose(embed_dim,pow(2, ceil(log2(vocab_size)))), seq_len,pow(2, ceil(log2(vocab_size))),  embed_dim);

    auto u1 = random_vec(ceilLog2(seq_len)); /*随机向量*/
    auto u2 = random_vec(ceilLog2(embed_dim)); /*随机向量*/
    auto ud = random_vec(ceilLog2(pow(2, ceil(log2(vocab_size)))));

    auto claim = GD_outputnorm_out.multi_dim_me({u1, u2}, {seq_len, embed_dim});/*MLE*/

    //auto final_claim = zkip(claim, Y.partial_me(u1,seq_len, pow(2, ceil(log2(vocab_size)))), GD_logistin.transpose(seq_len, pow(2, ceil(log2(vocab_size)))).partial_me(u2, seq_len, 1), ud, proof);  
    auto final_claim = zkip(claim,GD_logistin.partial_me(u1, seq_len, pow(2, ceil(log2(vocab_size)))), lm_weight_weight_padded.transpose(embed_dim,pow(2, ceil(log2(vocab_size)))).partial_me(u2,embed_dim, 1),  ud, proof);  

/*	损失对MLP输出 H 的梯度*/
   auto rmsnorm_weight = create_weight(
        workdir + "/" + "model.norm.weight-pp.bin",
        workdir + "/" + "model-norm-weight-int.bin",
        workdir + "/" + "model-norm-weight-commitment.bin",
        1, embed_dim
    );    
    FrTensor rms_inv_temp = FrTensor::from_int_bin("GD_rms_inv_temp_for_output.bin");


    Rescaling rs1(1 << 16), rs2(1 << 16);

    zkFC g = zkFC(1, embed_dim, rmsnorm_weight.weight);
    auto g_inv_rms = g(rms_inv_temp);

    auto g_inv_rms_ = rs1(g_inv_rms);

    auto GD_MLP_32_out = g_inv_rms_ * GD_outputnorm_out;

    auto v0 = ceilLog2(seq_len);
    auto v1 = ceilLog2(embed_dim);

    hadamard_product_sumcheck(g_inv_rms_, GD_outputnorm_out, random_vec(ceilLog2(GD_MLP_32_out.size)), random_vec(ceilLog2(GD_MLP_32_out.size)));
    rs1.prove(g_inv_rms, g_inv_rms_);    

    verifyWeightClaim(rmsnorm_weight, g.prove(rms_inv_temp, g_inv_rms)[0]);
    rms_inv_temp.~FrTensor();
    g_inv_rms.~FrTensor();
    g_inv_rms_.~FrTensor();

    cout << "GD for MLP_32_out successfully verified!" << endl; 


 		/*损失对自注意力输出 A_{\mathrm{out}}的梯度*/
    uint hidden_dim = 11008;


    auto up_proj = create_weight(
        workdir + "/mlp.up_proj.weight-pp.bin",
        workdir + "/" + "layer-31" + "-mlp.up_proj.weight-int.bin",
        workdir + "/" + "layer-31" + "-mlp.up_proj.weight-commitment.bin",
        embed_dim,
        hidden_dim
    );

    auto gate_proj = create_weight(
        workdir + "/mlp.gate_proj.weight-pp.bin",
        workdir + "/" + "layer-31" + "-mlp.gate_proj.weight-int.bin",
        workdir + "/" + "layer-31" + "-mlp.gate_proj.weight-commitment.bin",
        embed_dim,
        hidden_dim
    );

    auto down_proj = create_weight(
        workdir + "/mlp.down_proj.weight-pp.bin",
        workdir + "/" + "layer-31" + "-mlp.down_proj.weight-int.bin",
        workdir + "/" + "layer-31" + "-mlp.down_proj.weight-commitment.bin",
        hidden_dim,
        embed_dim
    );
    // cout << "up_proj" <<up_proj.weight.size <<endl; 
    // cout << "gate_proj" <<gate_proj.weight.size <<endl; 
    // cout << "down_proj" <<down_proj.weight.size <<endl; 

    zkFC up_layer(embed_dim, hidden_dim, up_proj.weight);
    zkFC gate_layer(embed_dim, hidden_dim, gate_proj.weight);
    zkFC down_layer(hidden_dim, embed_dim, down_proj.weight);

    Rescaling up_rescale(1 << 16);
    Rescaling gate_rescale(1 << 20);
    Rescaling hidden_rescale(1 << 16);
    Rescaling down_rescale(1 << 16);

    FrTensor swiglu_values = FrTensor::from_int_bin("swiglu-table.bin");
    FrTensor swiglu_gradient_values = FrTensor::from_int_bin("swiglu-gradient-table.bin");

    tLookupRangeMapping swiglu(-(1 << 21), 1 << 22, swiglu_values);
    tLookupRangeMapping swiglu_gradient(-(1 << 21), 1 << 22, swiglu_gradient_values);


    FrTensor Aout = FrTensor::from_int_bin(input_file_name);
    auto up_out = up_layer(Aout);
    auto up_out_ = up_rescale(up_out);
//    cout << "up_out" <<up_out.size<< endl;
//     cout << "up_proj" <<up_proj.weight.size<< endl; 

    auto gate_out = gate_layer(Aout);
    auto gate_out_ = gate_rescale(gate_out);
    auto p = swiglu(gate_out_);
    auto p_gradient = swiglu_gradient(gate_out_);

    auto &swiglu_out = p.first, &swiglu_m = p.second;
    auto &swiglu_gradient_out = p_gradient.first, &swiglu_gradient_m = p_gradient.second;

    auto temp_rand_1 = random_vec(3);
    auto swiglu_u = random_vec(ceilLog2(seq_len * hidden_dim));
    auto swiglu_v = random_vec(ceilLog2(seq_len * hidden_dim));
    vector<Polynomial> swiglu_proof, swiglu_gradient_out_proof;
    swiglu.prove(gate_out_, swiglu_out, swiglu_m, temp_rand_1[0], temp_rand_1[1], temp_rand_1[2], swiglu_u, swiglu_v, swiglu_proof);
    swiglu_gradient.prove(gate_out_, swiglu_gradient_out, swiglu_gradient_m, temp_rand_1[0], temp_rand_1[1], temp_rand_1[2], swiglu_u, swiglu_v, swiglu_gradient_out_proof);
    cout << "SwiGLU proof complete." << endl;
    vector<Polynomial> GD_swiglu_out_proof,gate_gradient_out_proof,up_gradient_out_proof;
    
    auto down_proj_weight_padded = pad_to_power_of_two(down_proj.weight,hidden_dim, embed_dim );
    uint pad_hidden_dim=pow(2, ceil(log2(hidden_dim)));
    auto GD_swiglu_out = FrTensor::matmul(GD_MLP_32_out, down_proj_weight_padded.transpose(pad_hidden_dim,embed_dim), seq_len, embed_dim, pad_hidden_dim);
   
    auto gate_gradient_in = swiglu_gradient_out * up_out_;

    auto gate_gradient_in_padded = pad_to_power_of_two(gate_gradient_in,seq_len, hidden_dim );
    auto gradient_gate = GD_swiglu_out * gate_gradient_in_padded;

    auto gate_proj_weight_padded = pad_to_power_of_two(gate_proj.weight, embed_dim,hidden_dim);
    auto gate_gradient_out = FrTensor::matmul(gradient_gate, gate_proj_weight_padded.transpose(embed_dim,pad_hidden_dim), seq_len, pad_hidden_dim, embed_dim);

    auto swiglu_out_padded = pad_to_power_of_two(swiglu_out,seq_len, hidden_dim );
    auto gradient_up = GD_swiglu_out * swiglu_out_padded;

    auto up_proj_weight_padded = pad_to_power_of_two(up_proj.weight,embed_dim, hidden_dim );
    auto up_gradient_out = FrTensor::matmul(gradient_up, up_proj_weight_padded.transpose(embed_dim,pad_hidden_dim), seq_len, pad_hidden_dim, embed_dim);
    

    auto post_att_normout = up_gradient_out + gate_gradient_out;

    auto u1_ = random_vec(ceilLog2(seq_len)); /*随机向量*/
    auto u2_ = random_vec(ceilLog2(embed_dim)); /*随机向量*/
    auto ud_ = random_vec(ceilLog2(pow(2, ceil(log2(hidden_dim)))));
    auto GD_swiglu_out_claim = GD_swiglu_out.multi_dim_me({u1_, ud_}, {seq_len, pad_hidden_dim});/*MLE*/

    auto GD_swiglu_out_final_claim = zkip(GD_swiglu_out_claim, GD_MLP_32_out.partial_me(u1_, seq_len, embed_dim), down_proj_weight_padded.transpose(pad_hidden_dim,embed_dim).partial_me(ud_,pad_hidden_dim, 1),  u2_, GD_swiglu_out_proof);  

    hadamard_product_sumcheck(swiglu_gradient_out, up_out_, random_vec(ceilLog2(gate_gradient_in.size)), random_vec(ceilLog2(gate_gradient_in.size)));

    hadamard_product_sumcheck(GD_swiglu_out, gate_gradient_in_padded, random_vec(ceilLog2(gradient_gate.size)), random_vec(ceilLog2(gradient_gate.size)));

    auto gate_gradient_out_claim = gate_gradient_out.multi_dim_me({u1_, u2_}, {seq_len, embed_dim});/*MLE*/
    auto gate_gradient_out_final_claim = zkip(gate_gradient_out_claim, gradient_gate.partial_me(u1_, seq_len, pad_hidden_dim), gate_proj_weight_padded.transpose(embed_dim,pad_hidden_dim).partial_me(u2_,embed_dim, 1),  ud_, gate_gradient_out_proof);  

    hadamard_product_sumcheck(GD_swiglu_out, swiglu_out_padded, random_vec(ceilLog2(gradient_up.size)), random_vec(ceilLog2(gradient_up.size)));

    auto up_gradient_out_claim = up_gradient_out.multi_dim_me({u1_, u2_}, {seq_len, embed_dim});/*MLE*/
    auto up_gradient_out_final_claim = zkip(up_gradient_out_claim, gradient_up.partial_me(u1_, seq_len, pad_hidden_dim), up_proj_weight_padded.transpose(embed_dim,pad_hidden_dim).partial_me(u2_,embed_dim, 1),  ud_, up_gradient_out_proof);  
    cout << "gd_post_att_normout." <<post_att_normout.size<< endl;

    post_att_normout.save("gd_post_att_normout.bin");
    cout << "GD to Att_32_out successfully verified!." << endl;


    return 0;
}


