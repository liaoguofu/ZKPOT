#include "hip/hip_runtime.h"
#include "zksoftmax.cuh"
#include "zkfc.cuh"
#include "fr-tensor.cuh"
#include "proof.cuh"
#include "commitment.cuh"
#include "rescaling.cuh"
#include <string>

int main(int argc, char *argv[])
{    
 //   hipSetDevice(3);
    // 从命令行参数获取 vocab_size 和 seq_len
    uint vocab_size = std::stoi(argv[1]);
    string one_hot_file = argv[2];
    string input_file_name = argv[3];
    uint seq_len = std::stoi(argv[4]);
    uint embed_dim = std::stoi(argv[5]);
    string workdir = argv[6];
    string layer_prefix = argv[7];

    auto lm_weight = create_weight(
        workdir + "/" + "lm_head.weight-pp.bin",
        workdir + "/" + "lm_head-weight-int.bin",
        workdir + "/" + "lm_head-weight-commitment.bin",
        seq_len, vocab_size
    );
    /*logits*/

    zkFC output_layer(4096, vocab_size, lm_weight.weight);

    FrTensor gd_post_att_normout = FrTensor::from_bin("gd_post_att_normout.bin");

   auto rmsnorm_weight = create_weight(
        workdir + "/" + "model.norm.weight-pp.bin",
        workdir + "/" + "model-norm-weight-int.bin",
        workdir + "/" + "model-norm-weight-commitment.bin",
        1, embed_dim
    );    
    FrTensor rms_inv_temp = FrTensor::from_int_bin("GD_rms_inv_temp_for_0_post_attention.bin");
    vector<Polynomial> proof;


    Rescaling rs1(1 << 16), rs2(1 << 16);

    zkFC g = zkFC(1, embed_dim, rmsnorm_weight.weight);
    auto g_inv_rms = g(rms_inv_temp);

    auto g_inv_rms_ = rs1(g_inv_rms);

    auto GD_att_out = g_inv_rms_ * gd_post_att_normout;

    auto v0 = ceilLog2(seq_len);
    auto v1 = ceilLog2(embed_dim);

    hadamard_product_sumcheck(g_inv_rms_, GD_att_out, random_vec(ceilLog2(GD_att_out.size)), random_vec(ceilLog2(GD_att_out.size)));
    rs1.prove(g_inv_rms, g_inv_rms_);    

    verifyWeightClaim(rmsnorm_weight, g.prove(rms_inv_temp, g_inv_rms)[0]);

    FrTensor softmax_out = FrTensor::from_bin("layer-31_softmax_out.bin");

    auto GD_att_v_out = FrTensor::matmul( softmax_out.transpose(seq_len,seq_len),GD_att_out, seq_len,seq_len, embed_dim);



    FrTensor initial_A = FrTensor::from_int_bin("initial_A.bin");
    FrTensor initial_B = FrTensor::from_int_bin("initial_B.bin");
    FrTensor attn_input = FrTensor::from_int_bin("attn_input.bin");

    auto BX = FrTensor::matmul(attn_input, initial_B, seq_len,embed_dim, 8);
    auto GD_B = FrTensor::matmul(BX.transpose(seq_len,8),GD_att_v_out,  8,seq_len, embed_dim);

    auto u1 = random_vec(ceilLog2(seq_len)); /*随机向量*/
    auto u2 = random_vec(ceilLog2(embed_dim)); /*随机向量*/
    auto u3 = random_vec(ceilLog2(8)); /*随机向量*/
    auto ud = random_vec(ceilLog2(pow(2, ceil(log2(vocab_size)))));

    auto claim = GD_att_v_out.multi_dim_me({u1, u2}, {seq_len, embed_dim});/*MLE*/

    auto final_claim = zkip(claim,softmax_out.transpose(seq_len,seq_len).partial_me(u1, seq_len, seq_len), GD_att_out.partial_me(u2,embed_dim, 1),  u1, proof);  
   
    cout << "GD to Att_32_V_out successfully verified!." << endl;
    auto BX_claim = BX.multi_dim_me({u1, u3}, {seq_len, 8});/*MLE*/
    auto BX_final_claim = zkip(BX_claim,attn_input.partial_me(u1, seq_len, embed_dim), initial_B.partial_me(u3,8, 1),  u2, proof);  
   
    auto GD_B_claim = GD_B.multi_dim_me({u3, u2}, {8, embed_dim});/*MLE*/
    auto GD_B_final_claim = zkip(GD_B_claim,BX.transpose(seq_len,8).partial_me(u3, 8, seq_len), GD_att_v_out.partial_me(u2,embed_dim, 1),  u1, proof);     
    cout << "GD to A successfully verified!." << endl;

    auto temp = FrTensor::matmul(attn_input.transpose(seq_len,embed_dim), GD_att_v_out, embed_dim,seq_len, embed_dim);
    auto GD_A = FrTensor::matmul(temp,initial_A.transpose(8,embed_dim),  embed_dim,embed_dim, 8);

    auto temp_claim = temp.multi_dim_me({u2, u2}, {embed_dim, embed_dim});/*MLE*/
    auto temp_final_claim = zkip(temp_claim,attn_input.transpose(seq_len,embed_dim).partial_me(u2, embed_dim, seq_len), GD_att_v_out.partial_me(u2,embed_dim, 1),  u1, proof);  
   
    auto GD_A_claim = GD_A.multi_dim_me({u2, u3}, {embed_dim, 8});/*MLE*/
    auto GD_A_final_claim = zkip(GD_A_claim,temp.partial_me(u2, embed_dim, embed_dim), initial_A.transpose(8,embed_dim).partial_me(u3,8, 1),  u2, proof);  
   
    cout << "GD to B successfully verified!." << endl;


    return 0;
}


